#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <math.h>
#include <string>
#include <algorithm>
#include <nvml.h>

#define CHECK_CUDA_ERROR(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "%s:%d: CUDA error %d: %s\n", __FILE__, __LINE__, result, hipGetErrorString(result)); \
        exit(1); \
    } \
} while (0)

#define CHECK_CUFFT_ERROR(call) \
do { \
    hipfftResult result = call; \
    if (result != HIPFFT_SUCCESS) { \
        fprintf(stderr, "%s:%d: cuFFT error %d\n", __FILE__, __LINE__, result); \
        exit(1); \
    } \
} while (0)

int main(int argc, char **argv) {

    long long num_iterations, num_inner_iterations;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    fprintf(stderr,"CUDA version: %d.%d\n", CUDART_VERSION / 1000, (CUDART_VERSION % 100) / 10);
    fprintf(stderr,"GPU: %s\n", prop.name);
    fprintf(stderr,"Driver compute compatibility: %d.%d\n", prop.major, prop.minor);
    // Initialize NVML library
    nvmlReturn_t result = nvmlInit();
    if (result != NVML_SUCCESS) {
        fprintf(stderr,"Failed to initialize NVML library: %s\n", nvmlErrorString(result));
        return 1;
    }
    char version_str[NVML_DEVICE_PART_NUMBER_BUFFER_SIZE+1];
    nvmlReturn_t retval = nvmlSystemGetDriverVersion(version_str, NVML_DEVICE_PART_NUMBER_BUFFER_SIZE);
    if (retval != NVML_SUCCESS) {
        fprintf(stderr, "%s\n",nvmlErrorString(retval));
        return 1;
    }
    fprintf(stderr,"Driver version: %s\n", version_str);

    num_iterations = 10;
    num_inner_iterations = 10000;
    int ntrials = 1;
    long long nffts[ntrials] = {1L<<7};
    std::string* description = new std::string[ntrials]{"2^7"};

    for (int i = 0; i < ntrials; i++){
        long long n = nffts[i];
        fprintf(stderr,"**************************************\n");
        fprintf(stderr,"N-point FFT: %lld (%s)\n", nffts[i], description[i].c_str());
        fprintf(stderr,"Number of iterations: %lld \n", num_iterations);


        int batch = 1;
        int rank = 1;
        long long nembed[1] = {n};
        int istride = 1;
        int ostride = 1;
        long long idist = n;
        long long odist = n;
        long long inembed[1] = {n};
        long long onembed[1] = {n};
        hipfftHandle forward_plan;
        hipEvent_t start, stop;
        float elapsed_time, inner_elapsed_time;
        float *input_data, *output_data;
        hipfftComplex *fft_data;
        float *host_input_data;
        float mean_time, median_time;
        size_t work_size;

        float input_size_gb = n*4.0/1e9;
        fprintf(stderr,"Input float array size: %lf GB \n", input_size_gb);
        float output_size_gb = n*8.0/1e9;
        fprintf(stderr,"Output complex array size: %lf GB \n", output_size_gb);


        // Allocate memory on host
        host_input_data = (float*) malloc(n * batch * sizeof(float));

        // Initialize input data on host
        srand(time(NULL));
        for (long int k = 0; k < n * batch; k++) {
            host_input_data[k] = (float) rand() / RAND_MAX;
        }
        //get size estimate
        hipfftResult result = hipfftEstimate1d(n, HIPFFT_R2C, batch, &work_size);
        float work_size_gb = work_size/1.0e9;
        fprintf(stderr,"Work size estimate: %lf GB\n", work_size_gb);
        fprintf(stderr, "Total size estimate: %lf GB\n", input_size_gb + output_size_gb + work_size_gb);

        // Allocate memory on device
        CHECK_CUDA_ERROR(hipMalloc((void**) &input_data, n * batch * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc((void**) &fft_data, n * batch * sizeof(hipfftComplex)));
        CHECK_CUDA_ERROR(hipMalloc((void**) &output_data, n * batch * sizeof(float)));

        // Create FFT plan
        CHECK_CUFFT_ERROR(hipfftCreate(&forward_plan));
        CHECK_CUFFT_ERROR(hipfftMakePlanMany64(forward_plan, rank, nembed, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch, &work_size));


        // Copy input data to device
        CHECK_CUDA_ERROR(hipMemcpy(input_data, host_input_data, n * batch * sizeof(float), hipMemcpyHostToDevice));

        mean_time = 0.0;

        //Calculate median time
        float times[num_iterations];
        for (int iter = 0; iter < num_iterations; iter++) {
            elapsed_time = 0.0;
            inner_elapsed_time = 0.0;
            for (int iter = 0; iter < num_inner_iterations; iter++) {
                CHECK_CUDA_ERROR(hipEventCreate(&start));
                CHECK_CUDA_ERROR(hipEventCreate(&stop));
                CHECK_CUDA_ERROR(hipEventRecord(start, 0));

                CHECK_CUFFT_ERROR(hipfftExecR2C(forward_plan, input_data, fft_data));


                CHECK_CUDA_ERROR(hipEventRecord(stop, 0));
                CHECK_CUDA_ERROR(hipEventSynchronize(stop));
                CHECK_CUDA_ERROR(hipEventElapsedTime(&inner_elapsed_time, start, stop));
                elapsed_time += inner_elapsed_time;
            }
            mean_time += elapsed_time;    
            times[iter] = elapsed_time;    
        }

        std::sort(times, times + num_iterations);
        if (num_iterations % 2 == 0) {
            median_time = (times[num_iterations / 2 - 1] + times[num_iterations / 2]) / 2.0;
        } else {
            median_time = times[num_iterations / 2];
        }


        mean_time = mean_time / num_iterations;

        fprintf(stderr,"Mean time: %f ms\n", mean_time);
        fprintf(stderr,"Median time: %f ms\n", median_time);

        // Free memory
        free(host_input_data);
        CHECK_CUDA_ERROR(hipFree(input_data));
        CHECK_CUDA_ERROR(hipFree(output_data));
        CHECK_CUFFT_ERROR(hipfftDestroy(forward_plan));
    }

    return 0;
}